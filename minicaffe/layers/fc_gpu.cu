#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#include "fc_layer.h"
#include "../seqnet.h"
#include "../util.h"
#include "../errors.h"


__global__ void mysgemm(int m, int n, int k, const float alpha, const float *A, const float *B, const float beta, float* C) {

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < n)
    {
        float tmp = beta * C[row*n+col];
        for(int i = 0; i < k; i++)
        {
            tmp += alpha * A[row*k+i]*B[i*n+col];
        }
        C[row * n + col] = tmp;
    }
}

void basicSgemm(int m, int n, int k, float alpha, const float *A, const float *B, float beta, float *C)
{
    const unsigned int BLOCK_SIZE = 512;

    dim3 dimGrid( (n-1)/BLOCK_SIZE + 1, (m-1)/BLOCK_SIZE+1, 1 );
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);

    mysgemm<<<dimGrid, dimBlock>>>(m, n, k, alpha, A, B, beta, C);
}

__global__ void VecAdd(int n, const float *A, const float *B, float* C) {

    int localIdx = threadIdx.x + blockDim.x * blockIdx.x;
    if (localIdx < n)
    {
        C[localIdx] = A[localIdx] + B[localIdx];
    } 
}


void basicVecAdd( float *A,  float *B, float *C, int n)
{

    const unsigned int BLOCK_SIZE = 512;

    dim3 DimGrid((n - 1) / BLOCK_SIZE + 1, 1, 1);
    dim3 DimBlock(BLOCK_SIZE, 1, 1);
    VecAdd<<<DimGrid,DimBlock>>>(n, A, B, C);
}

void FCLayer::infer_gpu(std::vector<Blob*> left_blobs, std::vector<Blob*> right_blobs)
{
    // treat _data as a 2D matrix
    // right = left * weight
    float *in_h, *out_h, *in_d, *out_d, *weight_h, *weight_d, *bias_d;
    int num_in, num_out, num_weight;
    hipError_t cuda_ret;

    num_in = M_ * K_;
    num_out = M_ * N_;
    num_weight = K_ * N_;

    in_h = left_blobs[0]->_data;
    out_h = (float*)malloc(num_out * sizeof(float));
    weight_h = weight;

    hipMalloc((void**)&in_d, num_in * sizeof(float));
    hipMalloc((void**)&out_d, num_out * sizeof(float));
    hipMalloc((void**)&weight_d, num_weight * sizeof(float));
    hipMalloc((void**)&bias_d, N_ * sizeof(float));

    hipDeviceSynchronize();

    hipMemcpy(in_d, in_h, num_in * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out_h, num_out * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(weight_d, weight_h, num_weight * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(bias_d, bias, N_ * sizeof(float), hipMemcpyHostToDevice);

    basicSgemm(M_, N_, K_, 1, in_d, weight_d, 0, out_d);

    hipDeviceSynchronize();

    // bias
    if (bias_term)
    {
        for (int b = 0; b < M_; b++)
        {
            basicVecAdd(out_d + b * N_, bias_d, out_d + b * N_, N_);
            hipDeviceSynchronize();
        }
    }

    hipMemcpy(out_h, out_d, num_out * sizeof(float), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    if (right_blobs[0]->_data) free(right_blobs[0]->_data);
    right_blobs[0]->_data = out_h;

    hipFree(in_d);
    hipFree(out_d);
    hipFree(weight_d);
    hipFree(bias_d);
}

void FCLayer::bp_gpu(std::vector<Blob*> lefts, std::vector<Blob*> rights)
{
    Blob* left = lefts[0];
    Blob* right = rights[0];

    // weight
    // we first transpose left from MxK --> KxM
    float* leftT = new float[K_ * M_];
    for (int row = 0; row < M_; row++)
    {
        for (int col = 0; col < K_; col++)
        {
            leftT[col * M_ + row] = left->_data[row * K_ + col];
        }
    }

    float *left_d, *left_h, *left_t_h, *left_t_d, *bias_h, *bias_d, *right_h, *right_d, *weight_h, *weight_d;
    int num_left, num_right, num_weight, num_bias;
    hipError_t cuda_ret;

    num_left = M_ * K_;
    num_right = M_ * N_;
    num_weight = K_ * N_;
    num_bias = N_;

    left_h = left->_data;
    left_t_h = leftT;
    bias_h = bias;
    weight_h = weight;
    right_h = right->_data;

    hipMalloc((void**)&left_d, num_left * sizeof(float));
    hipMalloc((void**)&right_d, num_right * sizeof(float));
    hipMalloc((void**)&weight_d, num_weight * sizeof(float));
    hipMalloc((void**)&bias_d, num_bias * sizeof(float));
    hipMalloc((void**)&left_t_d, num_left * sizeof(float));
    
    hipMemcpy(left_d, left_h, num_left * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(right_d, right_h, num_right * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(weight_d, weight_h, num_weight * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(bias_d, bias_h, num_bias * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(left_t_d, left_t_h, num_left * sizeof(float), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    basicSgemm(K_, N_, M_, 1, left_t_d, right_d, 1, weight_d);

    // bias
    if (bias_term)
    {
        for (int b = 0; b < M_; b++)
        {
            basicVecAdd(right_d + b * N_, bias_d, bias_d, N_);
            hipDeviceSynchronize();
        }
    }

    cuda_ret = hipDeviceSynchronize();
    {
        if (cuda_ret != hipSuccess)
            printf("errors occured!\n");
    }
    basicSgemm(M_, K_, N_, 1, right_d, weight_d, 0, left_d);
    hipDeviceSynchronize();

    cuda_ret = hipMemcpy(left_h, left_d, num_left * sizeof(float),
                          hipMemcpyDeviceToHost);
    if (cuda_ret != hipSuccess)
    {
        print_err_str(ZERO_DIM);
        exit(ZERO_DIM);
    }

    hipDeviceSynchronize();

    if (lefts[0]->_data) free(lefts[0]->_data);
    lefts[0]->_data = left_h;

    hipFree(left_d);
    hipFree(left_t_d);
    hipFree(right_d);
    hipFree(weight_d);
    hipFree(bias_d);

    delete[] leftT;
}
