#include "hip/hip_runtime.h"
#include "relu_layer.h"
#include <assert.h>

__global__ void relu_infer(float *out,float *in, unsigned size){
    unsigned idx=blockDim.x*blockIdx.x+threadIdx.x;
    if(idx<size){
        out[idx]=max(0.0,in[idx]);
    }
}


void ReluLayer::infer_gpu(vector<Blob *> lefts, vector<Blob *> rights) {
    rights[0]->reset();
    unsigned threads=256;
    int num_ele=lefts[0]->get_ele_num();

    float *in_h,*out_h,*in_d,*out_d;
    hipError_t cuda_ret;

    in_h=lefts[0]->_data;
    out_h=(float*)malloc(num_ele*sizeof(float));
    cuda_ret = hipMalloc((void**)&in_d, num_ele * sizeof(float));
    assert(cuda_ret == hipSuccess);
    cuda_ret = hipMalloc((void**)&out_d, num_ele * sizeof(float));
    if(cuda_ret == hipSuccess);
    hipDeviceSynchronize();

    cuda_ret = hipMemcpy(in_d, in_h, num_ele * sizeof(float),
                          hipMemcpyHostToDevice);
    assert(cuda_ret == hipSuccess);
    cuda_ret = hipMemset(out_d, 0, num_ele * sizeof(float));
    assert(cuda_ret == hipSuccess);
    hipDeviceSynchronize();

    dim3 dim_grid, dim_block;
    dim_block.x=threads;
    dim_block.y=dim_block.z=1;
    dim_grid.x=((num_ele-1)/threads+1);
    dim_grid.y=1;
    dim_grid.z=1;
    relu_infer<<<dim_grid,dim_block>>>(out_d,in_d,num_ele);
    cuda_ret = hipDeviceSynchronize();
    assert(cuda_ret == hipSuccess);

    cuda_ret = hipMemcpy(out_h, out_d, num_ele * sizeof(float),
                          hipMemcpyDeviceToHost);
    assert(cuda_ret == hipSuccess);
    hipDeviceSynchronize();

    delete[] rights[0]->_data;
    rights[0]->_data=out_h;
    hipFree(in_d);
    hipFree(out_d);

}

